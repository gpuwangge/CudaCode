//https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial01/


#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

using namespace std;

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Handling arbitrary vector size
    if (tid < n){
        out[tid] = a[tid] + b[tid];
    }
}

int main(){
	//Initialize Host Memory
    float *a, *b, *out; 
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }
	
	//Initialize Device Memory
	float *d_a, *d_b, *d_out; 
	hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);
	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    //Executing kernel 
	int block_size = 256;
    int grid_size = ((N + block_size) / block_size);
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);
	
	//Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
	
	//Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

	//for(int i = 0; i < 10; i++)
	//	cout<<"out is: "<<out[i]<<endl;
	//getchar();
	
	
	
	//Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
	//Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
