#include "hip/hip_runtime.h"
//CUDA by example(pdf), page 195
////////////////////////////
//本例展示了向量点积的高级原子实现
//kernel前半部分展示了(线程间)共享内存的用法；后半部分展示了归约算法
//所有计算都是使用GPU算的
////////////////////////////

#include <iostream>
#include <assert.h>
#include "book.h"
#include "lock.h"

using namespace std;

#define imin(a,b) (a<b?a:b)
const int N = 33 * 1024;
const int threadsPerBlock = 256; //this is blockDim.x
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1)/threadsPerBlock);//this is gridDim.x

__global__ void vector_dot(Lock lock, float *out, float *a, float *b) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//int cacheIndex = threadIdx.x;
	
	float tmp = 0;
	while(tid < N){
		tmp += a[tid] * b[tid];
		tid += threadsPerBlock * blocksPerGrid;//blockDim.x * gridDim.x; same effect
	}
	
	cache[threadIdx.x] = tmp;
	
	__syncthreads();
	
	int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;		
	}
	
	if(threadIdx.x == 0){
		//out[blockIdx.x] = cache[0];//这一句是老code
		//以下部分是新加入的原子锁代码
		lock.lock();
		*out += cache[0];
		lock.unlock();
	}
}

int main(){
	//Initialize Host Memory
    float *a, *b, out = 0; 
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    //out = (float*)malloc(sizeof(float) * blocksPerGrid);
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }
	
	//Initialize Device Memory
	float *d_a, *d_b, *d_out; 
	hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float));
	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_out, &out, sizeof(float), hipMemcpyHostToDevice);//新加入的原子锁相关code

    //Executing kernel 
	Lock lock;//新加入的原子锁相关code
    vector_dot<<<blocksPerGrid,threadsPerBlock>>>(lock, d_out, d_a, d_b);
	
	//Transfer data back to host memory
    hipMemcpy(&out, d_out, sizeof(float), hipMemcpyDeviceToHost);
	
	//finish the result in CPU
	//result = 0;
	//for(int i = 0; i < blocksPerGrid; i++) result += out[i];
	printf("out = %f\n", out);
	
	//Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
	//Deallocate host memory
    free(a); 
    free(b); 
    //free(out);
}
