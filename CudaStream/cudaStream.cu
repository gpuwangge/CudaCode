#include "hip/hip_runtime.h"
//CUDA by example(pdf), page 139
////////////////////////////
//本例展示了Stream,可以使GPU实现多任务
////////////////////////////


#include <iostream>
#include <assert.h>

using namespace std;

#define N2 (1024*1024)
#define FULL_DATA_SIZE (N2*20)
__global__ void kernel_stream(int *a, int *b, int *c){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < N2){
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int main(){
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);

	if(!prop.deviceOverlap){
		cout<<"The divice will not handle overlaps, so no speed up from streams."<<endl;
		return;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	/*Init streams*/
	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	/*Create identical stes of GPU buffers*/
	int *dev_a0, *dev_b0, *dev_c0;
	int *dev_a1, *dev_b1, *dev_c1;
	hipMalloc((void**)&dev_a0, N2 * sizeof(int));
	hipMalloc((void**)&dev_b0, N2 * sizeof(int));
	hipMalloc((void**)&dev_c0, N2 * sizeof(int));
	hipMalloc((void**)&dev_a1, N2 * sizeof(int));
	hipMalloc((void**)&dev_b1, N2 * sizeof(int));
	hipMalloc((void**)&dev_c1, N2 * sizeof(int));

	/*Allocate page-locked memory for stream*/
	int *host_a, *host_b, *host_c;
	hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	for(int i = 0; i < FULL_DATA_SIZE; i++){
		host_a[i] = rand();
		host_b[i] = rand();
	}

	
	for(int i = 0; i < FULL_DATA_SIZE; i += N2 * 2){
		///*Stream0*/
		//hipMemcpyAsync(dev_a0, host_a + i, N2 * sizeof(int), hipMemcpyHostToDevice, stream0);
		//hipMemcpyAsync(dev_b0, host_b + i, N2 * sizeof(int), hipMemcpyHostToDevice, stream0);
		//kernel_stream<<<N2/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
		//hipMemcpyAsync(host_c + i, dev_c0, N2 * sizeof(int), hipMemcpyDeviceToHost, stream0);

		///*Stream1*/
		//hipMemcpyAsync(dev_a1, host_a + i + N2, N2 * sizeof(int), hipMemcpyHostToDevice, stream1);
		//hipMemcpyAsync(dev_b1, host_b + i + N2, N2 * sizeof(int), hipMemcpyHostToDevice, stream1);
		//kernel_stream<<<N2/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
		//hipMemcpyAsync(host_c + i + N2, dev_c1, N2 * sizeof(int), hipMemcpyDeviceToHost, stream1);

		/*Optimized version*/
		hipMemcpyAsync(dev_a0, host_a + i, N2 * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_a1, host_a + i + N2, N2 * sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dev_b0, host_b + i, N2 * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_b1, host_b + i + N2, N2 * sizeof(int), hipMemcpyHostToDevice, stream1);
		kernel_stream<<<N2/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
		kernel_stream<<<N2/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
		hipMemcpyAsync(host_c + i, dev_c0, N2 * sizeof(int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(host_c + i + N2, dev_c1, N2 * sizeof(int), hipMemcpyDeviceToHost, stream1);
	}

	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapseTime;
	hipEventElapsedTime(&elapseTime, start, stop);
	cout<<"It took "<<elapseTime/1000<<" Seconds."<<endl;

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipHostFree(dev_a0);
	hipHostFree(dev_b0);
	hipHostFree(dev_c0);
	hipHostFree(dev_a1);
	hipHostFree(dev_b1);
	hipHostFree(dev_c1);
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);
	return 0;
}
